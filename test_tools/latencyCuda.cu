#include "latencyCuda.h"


void init(int local_gpu,int* d_data){
    hipSetDevice(local_gpu);
    hipMalloc(&d_data,10*sizeof(int));
    int a[10] = {9,7,5,5,7,8,9,0,1,1};
    hipMemcpy(d_data,a, 10*sizeof(int), hipMemcpyHostToDevice);
}

void cudaDTH(void* dst,void* src){
    hipMemcpy(dst,src, 4, hipMemcpyDeviceToHost);
}
void cudaHTD(void* dst,void* src){
    hipMemcpy(dst,src, 4, hipMemcpyHostToDevice);
}