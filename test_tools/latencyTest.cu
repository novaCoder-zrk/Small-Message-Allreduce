
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <vector>
#include <pthread.h>
using namespace std;

#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
}

void outputLatencyMatrix(int numGPUs)
{
    int repeat=10000;
    vector<int *> buffers(numGPUs);
    vector<hipEvent_t> start(numGPUs);
    vector<hipEvent_t> stop(numGPUs);

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipMalloc(&buffers[d],8);
        cudaCheckError();
        hipEventCreate(&start[d]);
        cudaCheckError();
        hipEventCreate(&stop[d]);
        cudaCheckError();
    }

    vector<double> latencyMatrix(numGPUs*numGPUs);

    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {

            hipDeviceSynchronize();
            cudaCheckError();
            hipEventRecord(start[i]);

            for (int r=0; r<repeat; r++)
            {
                hipMemcpyPeerAsync(buffers[i],i,buffers[j],j,1);
            }

            hipEventRecord(stop[i]);
            hipDeviceSynchronize();
            cudaCheckError();

            float time_ms;
            hipEventElapsedTime(&time_ms,start[i],stop[i]);

            latencyMatrix[i*numGPUs+j]=time_ms*1e3/repeat;
        }
    }

    printf("   D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d ", j);
    }

    printf("\n");

    for (int i=0; i<numGPUs; i++)
    {
        printf("%6d ",i);

        for (int j=0; j<numGPUs; j++)
        {
            printf("%6.02f ", latencyMatrix[i*numGPUs+j]);
        }

        printf("\n");
    }

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipFree(buffers[d]);
        cudaCheckError();
        hipEventDestroy(start[d]);
        cudaCheckError();
        hipEventDestroy(stop[d]);
        cudaCheckError();
    }
}

vector<int *> buffers(8);
vector<hipStream_t> stream(8);
int* rootBuffer;

void* p2pMemCpy(void* arg) 
{
    
    int root = *(int*)arg;
    int d = *((int*)arg+1);
    int rank = *((int*)arg + 2);
    int repeat = *((int*)arg +3);
    hipSetDevice(rank);
    for(int r=0;r <repeat;r++){
        hipMemcpyPeerAsync(rootBuffer+d,root,buffers[d],rank,2,stream[d]);
    }
    return NULL;
}

void outputLatencyRoot(int root,vector<int> ranks,int numGPUs)
{
    int repeat=10000;


    hipEvent_t start;
    hipEvent_t stop;
    // root
    
    hipSetDevice(root);
    hipMalloc(&rootBuffer,32);
    hipEventCreate(&start);
    cudaCheckError();
    hipEventCreate(&stop);
    cudaCheckError();
    for(int d= 0;d < numGPUs;d++){
        hipStreamCreate(&stream[d]);
        cudaCheckError();
    }

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(ranks[d]);
        hipMalloc(&buffers[d],8);
        cudaCheckError();
        // cudaEventCreate(&start[d]);
        // cudaCheckError();
        // cudaEventCreate(&stop[d]);
        // cudaCheckError();
    }

    pthread_t threads[8];
    
     

    hipSetDevice(root);
    hipDeviceSynchronize();
    cudaCheckError();
    hipEventRecord(start);
    cudaCheckError();
    for(int d = 0;d < numGPUs;d++){
        int arg[] = {root, d, ranks[d], repeat};
        pthread_create(&threads[d],NULL,p2pMemCpy,arg);
    }

    for (int i = 0; i < numGPUs; i++) {
        pthread_join(threads[i], NULL);
    }
    hipSetDevice(root);
    cudaCheckError();
    hipEventRecord(stop);
    cudaCheckError();
    hipDeviceSynchronize();
    cudaCheckError();

    float time_ms;
    hipEventElapsedTime(&time_ms,start,stop);
    double latency = time_ms*1e3/repeat;
    printf("\n latency %6.02f ",latency);

}

int main(int argc,char *argv[])
{
    vector<int> ranks;
    for(int i = 1;i< argc;i++){
        int dev = *argv[i] - '0';
        ranks.push_back(dev);
    }
    //outputLatencyMatrix(ndev);

    
    outputLatencyRoot(0,ranks,ranks.size());
}