
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <vector>
#include <pthread.h>
using namespace std;

#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
}




vector<hipEvent_t> start(8);
vector<hipEvent_t> stop(8);
hipStream_t stream[8][2];
vector<int *> buffers(8);

float latency[8];
void* binaryMemCpy(void * arg)
{
    int rank1 = *(int*)arg;
    int rank2 = *((int*)arg+1);
    int repeat = *((int*)arg +2);
    hipSetDevice(rank1);
    

    hipDeviceSynchronize();
    hipEventRecord(start[rank1]);
    
    for(int r=0;r <repeat;r++){
       hipMemcpyPeerAsync(buffers[rank2],rank2,buffers[rank1]+1,rank1,1);
       hipMemcpyPeerAsync(buffers[rank1],rank1,buffers[rank2]+1,rank2,1);
        //cudaMemcpyPeerAsync(buffers[rank1],rank1,buffers[rank2],rank2,1,stream[rank1][0]);
        //cudaMemcpyPeerAsync(buffers[rank2],rank2,buffers[rank1],rank1,1,stream[rank1][1]);
    }
    hipEventRecord(stop[rank1]);
    cudaCheckError();
    hipDeviceSynchronize();
    float time_ms;
    hipEventElapsedTime(&time_ms,start[rank1],stop[rank1]);
    latency[rank1] = time_ms*1e3/repeat;
    return NULL;
}

void outputBiranyLatencyAll(vector<int> ranks1,vector<int> ranks2)
{
    int repeat=10000;
    
    for(int d= 0;d < 8;d++){
        hipSetDevice(d);
        hipStreamCreate(&stream[d][0]);
        hipStreamCreate(&stream[d][1]);
        hipEventCreate(&start[d]);
        hipEventCreate(&stop[d]);
        cudaCheckError();
    }

    for (int d=0; d< 8 ; d++)
    {
        hipSetDevice(d);
        hipMalloc(&buffers[d],8);
        cudaCheckError();
    }

    pthread_t threads[8];
    
    cudaCheckError();
    for(int i = 0;i < ranks1.size();i++){
        int arg[] = {ranks1[i], ranks2[i], repeat};
        pthread_create(&threads[i],NULL, binaryMemCpy, arg);
    }

    for (int i = 0; i < ranks1.size(); i++) {
        pthread_join(threads[i], NULL);
    }
    float max_latency = -1;
    for(int i = 0;i < ranks1.size();i++)
    {
        max_latency = max(max_latency, latency[ranks1[i]]);
    }

    
    printf("\n latency %6.02f ",max_latency);

}







int main(int argc,char *argv[])
{
    vector<int> ranks1;
    vector<int> ranks2;
    for(int i = 1;i< argc;i += 2){
        int dev;
        dev = *argv[i] - '0';
        ranks1.push_back(dev);
        dev = *argv[i+1] - '0';
        ranks2.push_back(dev);
    }
    for(int i = 0;i < ranks1.size();i++){
        printf("%d %d\t",ranks1[i],ranks2[i]);
    }
    //outputLatencyMatrix(ndev);
    outputBiranyLatencyAll(ranks1,ranks2);
    //outputLatencyRoot(0,ranks,ranks.size());
    //outputBinaryLatency(0,1);
}