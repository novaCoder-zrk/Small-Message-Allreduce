#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "nvmlwrap.h"

#define CHECK_CUDA_ERROR(call) \
do { \
  hipError_t result = call; \
  if (result != hipSuccess) { \
    fprintf(stderr, "%s:%d: CUDA error (%d): %s.\n", __FILE__, __LINE__, static_cast<int>(result), hipGetErrorString(result)); \
     \
  } \
} while(0)

void enableP2P(int numGPUs)
{
    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {
            int access;
            hipDeviceCanAccessPeer(&access,i,j);

            if (access)
            {
                hipDeviceEnablePeerAccess(j,0);
               
            }
        }
    }
}
int main()
{
    int access;
    hipSetDevice(0);
    for(int i = 1;i < 8;i++){
        hipDeviceCanAccessPeer(&access,0,i);
        printf("%s \n",access ? "access" : "not access");

        CHECK_CUDA_ERROR(hipDeviceEnablePeerAccess(i,0));
    }

}